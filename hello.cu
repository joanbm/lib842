
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {

	float * ha;
	float * da;

	ha = (float*) malloc(512*sizeof(float));
	hipMalloc((void**) &da, 512*sizeof(float));

	for (int i = 0; i < 512; i++) {
		ha[i] = i;
	}
	
	hipMemcpy(da, ha, 512*sizeof(float), hipMemcpyHostToDevice);
	
	return 0;
}
