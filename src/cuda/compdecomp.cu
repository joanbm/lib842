#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>


#include "sw842.h"

//#define CHUNK_SIZE 32768
#define CHUNK_SIZE 1024
#define THREADS_PER_BLOCK 32
#define STRLEN 32

__global__ void cuda842_decompress(uint64_t *in, unsigned int ilen, uint64_t *out);

#define CHECK_ERROR( err ) \
  if( err != hipSuccess ) { \
    printf("Error: %s\n", hipGetErrorString(err)); \
    exit( -1 ); \
  }

long long timestamp() {
	struct timeval te;
	gettimeofday(&te, NULL);
	long long ms = te.tv_sec * 1000LL + te.tv_usec/1000;
	return ms;
}

int nextMultipleOfChunkSize(unsigned int input) {
	unsigned int size = CHUNK_SIZE * THREADS_PER_BLOCK;
	return (input + (size-1)) & ~(size-1);
} 

int main( int argc, const char* argv[])
{
	uint8_t *inH, *compressedH, *decompressedH, *transposedH;
	uint64_t *inD, *compressedD, *decompressedD;
	inH = compressedH = decompressedH = NULL;
	unsigned int ilen, olen, dlen;
	ilen = olen = dlen = 0;
	long long timestart_comp, timeend_comp;
	long long timestart_decomp, timeend_decomp;
	hipError_t cuda_error;
	int count = 0;
	hipGetDeviceCount(&count);
  	printf(" %d CUDA devices found\n", count);
  	if(!count)
    		::exit(EXIT_FAILURE);


	if(argc <= 1) {
		ilen = STRLEN;
		olen = ilen * 2;
		dlen = ilen;
		inH = (uint8_t*) malloc(ilen);
		hipMalloc((void**) &inD, ilen);
		compressedH = (uint8_t*) malloc(olen);
		hipMalloc((void**) &compressedD, olen);
		decompressedH = (uint8_t*) malloc(dlen);
		hipMalloc((void**) &decompressedD, dlen);
		memset(inH, 0, ilen);
		hipMemset(inD, 0, ilen);
		memset(compressedH, 0, olen);
		hipMemset(compressedD, 0, olen);
		memset(decompressedH, 0, dlen);
		hipMemset(decompressedD, 0, dlen);

		uint8_t tmp[] = {0x30, 0x30, 0x31, 0x31, 0x32, 0x32, 0x33, 0x33, 0x34, 0x34, 0x35, 0x35, 0x36, 0x36, 0x37, 0x37, 0x38, 0x38, 0x39, 0x39, 0x40, 0x40, 0x41, 0x41, 0x42, 0x42, 0x43, 0x43, 0x44, 0x44, 0x45, 0x45};//"0011223344556677889900AABBCCDDEE";
		strncpy((char *) inH, (const char *) tmp, STRLEN);

	} else if (argc == 2) {
		FILE *fp;
		fp=fopen(argv[1], "r");
		fseek(fp, 0, SEEK_END);
		unsigned int flen = ftell(fp);
		ilen = flen;
		printf("original file length: %d\n", ilen);
		ilen = nextMultipleOfChunkSize(ilen);
		printf("original file length (padded): %d\n", ilen);
		olen = ilen * 2;
		dlen = ilen;
		fseek(fp, 0, SEEK_SET);

		inH = (uint8_t*) malloc(ilen);
		hipMalloc((void**) &inD, ilen);
		compressedH = (uint8_t*) malloc(olen);
		transposedH = (uint8_t*) malloc(olen);
		hipMalloc((void**) &compressedD, olen);
		decompressedH = (uint8_t*) malloc(dlen);
		hipMalloc((void**) &decompressedD, dlen);
		memset(inH, 0, ilen);
		hipMemset(inD, 0, ilen);
		memset(compressedH, 0, olen);
		hipMemset(compressedD, 0, olen);
		memset(decompressedH, 0, dlen);
		hipMemset(decompressedD, 0, dlen);


		if(!fread(inH, flen, 1, fp)) {
			fprintf(stderr, "FAIL: Reading file content to memory failed.\n");
		}
		fclose(fp);
	}

	if(ilen > CHUNK_SIZE) {
		printf("Using chunks of %d bytes\n", CHUNK_SIZE);
	
		timestart_comp = timestamp();
		#pragma omp parallel for
		for(int chunk_num = 0; chunk_num < ilen / CHUNK_SIZE; chunk_num++) {
			
			unsigned int chunk_olen = CHUNK_SIZE * 2;
			uint8_t* chunk_in = inH + (CHUNK_SIZE * chunk_num);
			uint8_t* chunk_out = compressedH + ((CHUNK_SIZE * 2) * chunk_num);
			
			sw842_compress(chunk_in, CHUNK_SIZE, chunk_out, &chunk_olen);

			for(int i = 0; i < 256; i++){
				memcpy(transposedH + (i*(CHUNK_SIZE*2)) + (chunk_num*8), chunk_out + (i*8), 8);
			}
		}
		timeend_comp = timestamp();




		cuda_error = hipMemcpy(compressedD, compressedH, olen, hipMemcpyHostToDevice);
		CHECK_ERROR(cuda_error);

		timestart_decomp = timestamp();

		printf("Threads per Block: %d\n", THREADS_PER_BLOCK );

		cuda842_decompress<<<(ilen / CHUNK_SIZE) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(compressedD, ilen, decompressedD);
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);

		timeend_decomp = timestamp();

		cuda_error = hipMemcpy(decompressedH, decompressedD, dlen, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);

		printf("Compression performance: %lld ms / %f MiB/s\n", timeend_comp - timestart_comp, (ilen / 1024 / 1024) / ((float) (timeend_comp - timestart_comp) / 1000));
		printf("Decompression performance: %lld ms / %f MiB/s\n", timeend_decomp - timestart_decomp, (ilen / 1024 / 1024) / ((float) (timeend_decomp - timestart_decomp) / 1000));


	} else {

		sw842_compress(inH, ilen, compressedH, &olen);
		printf("copying compressed data to device\n");
		cuda_error = hipMemcpy(compressedD, compressedH, olen, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);
        printf("starting with device-based decompression\n");
        cuda842_decompress<<<1,1>>>(compressedD, olen, decompressedD);
        printf("copying decompressed data back to the host\n");
		cuda_error = hipMemcpy(decompressedH, decompressedD, dlen, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);

	}
	
	if (memcmp(inH, decompressedH, ilen) == 0) {
		printf("Compression- and decompression was successful!\n");
	} else {
		fprintf(stderr, "FAIL: Decompressed data differs from the original input data.\n");
		FILE *fpIn, *fpOut;
		fpIn=fopen("original.bin", "w");
		fwrite(inH, ilen, 1, fpIn);
		fclose(fpIn);
		fpOut=fopen("decompressed.bin", "w");
		fwrite(decompressedH, dlen, 1, fpOut);
		fclose(fpOut);
		free(inH);
		free(compressedH);
		free(decompressedH);
		hipFree(inD);
		hipFree(compressedD);
		hipFree(decompressedD);
		return 0;
	}
	free(inH);
	free(compressedH);
	free(decompressedH);
	hipFree(inD);
	hipFree(compressedD);
	hipFree(decompressedD);

	printf("\n\n");
	return 0;
}
