#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>


#include "sw842.h"

//#define CHUNK_SIZE 32768
#define CHUNK_SIZE 1024
#define THREADS_PER_BLOCK 32
#define STRLEN 32

__global__ void cuda842_decompress(uint64_t *in, uint64_t *out, uint32_t num_chunks);

#define CHECK_ERROR( err ) \
  if( err != hipSuccess ) { \
    printf("Error: %s\n", hipGetErrorString(err)); \
    exit( -1 ); \
  }

long long timestamp() {
	struct timeval te;
	gettimeofday(&te, NULL);
	long long ms = te.tv_sec * 1000LL + te.tv_usec/1000;
	return ms;
}

int nextMultipleOfChunkSize(unsigned int input) {
	unsigned int size = CHUNK_SIZE * THREADS_PER_BLOCK;
	return (input + (size-1)) & ~(size-1);
} 

int row_major_from_column_major(int column_major, int width, int height) {
    int row = column_major % height;
    int column = column_major / height;
    return row * width + column;
}

int column_major_from_row_major(int row_major, int width, int height) {
    return row_major_from_column_major(row_major, height, width);
}

int main( int argc, const char* argv[])
{
	uint8_t *inH, *compressedH, *compressedTransH, *decompressedH;
	uint64_t *compressedD, *decompressedD;
	inH = compressedH = compressedTransH = decompressedH = NULL;
	unsigned int num_chunks;
	unsigned int ilen, olen, dlen;
	ilen = olen = dlen = 0;
	long long timestart_comp, timeend_comp;
	long long timestart_decomp, timeend_decomp;
	hipError_t cuda_error;
	int count = 0;
	hipGetDeviceCount(&count);
  	printf(" %d CUDA devices found\n", count);
  	if(!count)
    		::exit(EXIT_FAILURE);


	if(argc <= 1) {
		ilen = STRLEN;
		olen = ilen * 2;
		dlen = ilen;
		inH = (uint8_t*) malloc(ilen);
		compressedH = (uint8_t*) malloc(olen);
		compressedTransH = (uint8_t*) malloc(olen);
		decompressedH = (uint8_t*) malloc(dlen);
		memset(inH, 0, ilen);
		memset(compressedH, 0, olen);
		memset(compressedTransH, 0, olen);
		memset(decompressedH, 0, dlen);

		hipMalloc((void**) &compressedD, olen);
		hipMalloc((void**) &decompressedD, dlen);
		hipMemset(compressedD, 0, olen);
		hipMemset(decompressedD, 0, dlen);

		uint8_t tmp[] = {0x30, 0x30, 0x31, 0x31, 0x32, 0x32, 0x33, 0x33, 0x34, 0x34, 0x35, 0x35, 0x36, 0x36, 0x37, 0x37, 0x38, 0x38, 0x39, 0x39, 0x40, 0x40, 0x41, 0x41, 0x42, 0x42, 0x43, 0x43, 0x44, 0x44, 0x45, 0x45};//"0011223344556677889900AABBCCDDEE";
		strncpy((char *) inH, (const char *) tmp, STRLEN);

	} else if (argc == 2) {
		FILE *fp;
		fp=fopen(argv[1], "r");
		fseek(fp, 0, SEEK_END);
		unsigned int flen = ftell(fp);
		ilen = flen;
		printf("original file length: %d\n", ilen);
		ilen = nextMultipleOfChunkSize(ilen);
		printf("original file length (padded): %d\n", ilen);
		olen = ilen * 2;
		dlen = ilen;
		num_chunks = ilen / CHUNK_SIZE;
		fseek(fp, 0, SEEK_SET);

		inH = (uint8_t*) malloc(ilen);
		compressedH = (uint8_t*) malloc(olen);
		compressedTransH = (uint8_t*) malloc(olen);
		decompressedH = (uint8_t*) malloc(dlen);
		memset(inH, 0, ilen);
		memset(compressedH, 0, olen);
		memset(compressedTransH, 0, olen);
		memset(decompressedH, 0, dlen);

		hipMalloc((void**) &compressedD, olen);
		hipMalloc((void**) &decompressedD, dlen);
		hipMemset(compressedD, 0, olen);
		hipMemset(decompressedD, 0, dlen);


		if(!fread(inH, flen, 1, fp)) {
			fprintf(stderr, "FAIL: Reading file content to memory failed.\n");
		}
		fclose(fp);
	}

	if(ilen > CHUNK_SIZE) {
		printf("Using chunks of %d bytes\n", CHUNK_SIZE);

		uint32_t num_chunks = ilen / CHUNK_SIZE;
		uint64_t *compressedChunkPositions = (uint64_t*) malloc(sizeof(uint64_t) * num_chunks);
		uint32_t *compressedChunkSizes = (uint32_t*) malloc(sizeof(uint32_t) * num_chunks);
	
		timestart_comp = timestamp();
		#pragma omp parallel for
		for(uint32_t chunk_num = 0; chunk_num < num_chunks; chunk_num++) {
			
			uint32_t chunk_olen = CHUNK_SIZE * 2;
			uint8_t* chunk_in = inH + (CHUNK_SIZE * chunk_num);
			uint8_t* chunk_out = compressedH + ((CHUNK_SIZE * 2) * chunk_num);
			
			sw842_compress(chunk_in, CHUNK_SIZE, chunk_out, &chunk_olen);
			compressedChunkSizes[chunk_num] = chunk_olen;
		}
		timeend_comp = timestamp();

		uint64_t *compressedH64, *compressedTransH64;
		compressedH64 = (uint64_t*) compressedH;
		compressedTransH64 = (uint64_t*) compressedTransH;

		for(int i = 0; i < olen/8;i++) {
			compressedTransH64[column_major_from_row_major(i, (CHUNK_SIZE*2)/8 , num_chunks)] = compressedH64[i];
		}


		cuda_error = hipMemcpy(compressedD, compressedTransH, olen, hipMemcpyHostToDevice);
		CHECK_ERROR(cuda_error);

		timestart_decomp = timestamp();

		printf("Threads per Block: %d\n", THREADS_PER_BLOCK );

		cuda842_decompress<<<num_chunks / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(compressedD, decompressedD, num_chunks);
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);

		timeend_decomp = timestamp();

		cuda_error = hipMemcpy(decompressedH, decompressedD, dlen, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);

		printf("Compression performance: %lld ms / %f MiB/s\n", timeend_comp - timestart_comp, (ilen / 1024 / 1024) / ((float) (timeend_comp - timestart_comp) / 1000));
		printf("Decompression performance: %lld ms / %f MiB/s\n", timeend_decomp - timestart_decomp, (ilen / 1024 / 1024) / ((float) (timeend_decomp - timestart_decomp) / 1000));


	} else {

		sw842_compress(inH, ilen, compressedH, &olen);
		printf("copying compressed data to device\n");
		cuda_error = hipMemcpy(compressedD, compressedH, olen, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);
        printf("starting with device-based decompression\n");
        cuda842_decompress<<<1,1>>>(compressedD, decompressedD, 1);
        printf("copying decompressed data back to the host\n");
		cuda_error = hipMemcpy(decompressedH, decompressedD, dlen, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);

	}
	
	if (memcmp(inH, decompressedH, ilen) == 0) {
		printf("Compression- and decompression was successful!\n");
	} else {
		fprintf(stderr, "FAIL: Decompressed data differs from the original input data.\n");
	}
	free(inH);
	free(compressedH);
	free(compressedTransH);
	free(decompressedH);

	hipFree(compressedD);
	hipFree(decompressedD);

	printf("\n\n");
	return 0;
}
