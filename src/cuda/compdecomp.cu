#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>


#include "sw842.h"

#define THREADS_PER_BLOCK 32
#define STRLEN 32

__global__ void cuda842_decompress(uint64_t *in, uint64_t *out, uint32_t num_chunks);

#define CHECK_ERROR( err ) \
  if( err != hipSuccess ) { \
    printf("Error: %s\n", hipGetErrorString(err)); \
    exit( -1 ); \
  }

long long timestamp() {
	struct timeval te;
	gettimeofday(&te, NULL);
	long long ms = te.tv_sec * 1000LL + te.tv_usec/1000;
	return ms;
}

int nextMultipleOfChunkSize(unsigned int input) {
	unsigned int size = CHUNK_SIZE * THREADS_PER_BLOCK;
	return (input + (size-1)) & ~(size-1);
} 

int main( int argc, const char* argv[])
{
	#ifdef STRICT
	printf("Running in strict mode (i.e. fully compatible to the hardware-based nx842 unit).\n");
	#endif
	uint8_t *inH, *compressedH, *decompressedH;
	uint64_t *compressedD, *decompressedD;
	hipStream_t stream;
	hipStreamCreate(&stream);
	unsigned int ilen, olen, dlen;
	ilen = olen = dlen = 0;
	long long timestart_comp, timeend_comp;
	long long timestart_decomp, timeend_decomp;
	hipError_t cuda_error;
	int count = 0;
	hipGetDeviceCount(&count);
  	printf(" %d CUDA devices found\n", count);
  	if(!count)
    		::exit(EXIT_FAILURE);


	if(argc <= 1) {
		ilen = STRLEN;
		olen = ilen * 2;
		dlen = ilen;
		hipHostAlloc((void**) &inH, ilen, hipHostMallocPortable);
		hipHostAlloc((void**) &compressedH, olen, hipHostMallocPortable);
		hipHostAlloc((void**) &decompressedH, dlen, hipHostMallocPortable);
		memset(inH, 0, ilen);
		memset(compressedH, 0, olen);
		memset(decompressedH, 0, dlen);

		hipMalloc((void**) &compressedD, olen);
		hipMalloc((void**) &decompressedD, dlen);
		hipMemset(compressedD, 0, olen);
		hipMemset(decompressedD, 0, dlen);

		uint8_t tmp[] = {0x30, 0x30, 0x31, 0x31, 0x32, 0x32, 0x33, 0x33, 0x34, 0x34, 0x35, 0x35, 0x36, 0x36, 0x37, 0x37, 0x38, 0x38, 0x39, 0x39, 0x40, 0x40, 0x41, 0x41, 0x42, 0x42, 0x43, 0x43, 0x44, 0x44, 0x45, 0x45};//"0011223344556677889900AABBCCDDEE";
		strncpy((char *) inH, (const char *) tmp, STRLEN);

	} else if (argc == 2) {
		FILE *fp;
		fp=fopen(argv[1], "r");
		fseek(fp, 0, SEEK_END);
		unsigned int flen = ftell(fp);
		ilen = flen;
		printf("original file length: %d\n", ilen);
		ilen = nextMultipleOfChunkSize(ilen);
		printf("original file length (padded): %d\n", ilen);
		olen = ilen * 2;
		dlen = ilen;
		fseek(fp, 0, SEEK_SET);

		hipHostAlloc((void**) &inH, ilen, hipHostMallocPortable);
		hipHostAlloc((void**) &compressedH, olen, hipHostMallocPortable);
		hipHostAlloc((void**) &decompressedH, dlen, hipHostMallocPortable);
		memset(inH, 0, ilen);
		memset(compressedH, 0, olen);
		memset(decompressedH, 0, dlen);

		hipMalloc((void**) &compressedD, olen);
		hipMalloc((void**) &decompressedD, dlen);
		hipMemset(compressedD, 0, olen);
		hipMemset(decompressedD, 0, dlen);


		if(!fread(inH, flen, 1, fp)) {
			fprintf(stderr, "FAIL: Reading file content to memory failed.\n");
		}
		fclose(fp);
	}

	if(ilen > CHUNK_SIZE) {
		printf("Using chunks of %d bytes\n", CHUNK_SIZE);

		uint32_t num_chunks = ilen / CHUNK_SIZE;
		uint64_t *compressedChunkPositions = (uint64_t*) malloc(sizeof(uint64_t) * num_chunks);
		uint32_t *compressedChunkSizes = (uint32_t*) malloc(sizeof(uint32_t) * num_chunks);
	
		timestart_comp = timestamp();
		#pragma omp parallel for
		for(uint32_t chunk_num = 0; chunk_num < num_chunks; chunk_num++) {
			
			uint32_t chunk_olen = CHUNK_SIZE * 2;
			uint8_t* chunk_in = inH + (CHUNK_SIZE * chunk_num);
			uint8_t* chunk_out = compressedH + ((CHUNK_SIZE * 2) * chunk_num);
			
			sw842_compress(chunk_in, CHUNK_SIZE, chunk_out, &chunk_olen);
			compressedChunkSizes[chunk_num] = chunk_olen;
		}
		timeend_comp = timestamp();

		cuda_error = hipMemcpy(compressedD, compressedH, olen, hipMemcpyHostToDevice);
		CHECK_ERROR(cuda_error);

		timestart_decomp = timestamp();

		printf("Threads per Block: %d\n", THREADS_PER_BLOCK );

		cuda842_decompress<<<num_chunks / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(compressedD, decompressedD, num_chunks);
		hipDeviceSynchronize();
		cuda_error = hipGetLastError();
		CHECK_ERROR(cuda_error);

		timeend_decomp = timestamp();

		cuda_error = hipMemcpy(decompressedH, decompressedD, dlen, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);

		printf("Compression performance: %lld ms / %f MiB/s\n", timeend_comp - timestart_comp, (ilen / 1024 / 1024) / ((float) (timeend_comp - timestart_comp) / 1000));
		printf("Decompression performance: %lld ms / %f MiB/s\n", timeend_decomp - timestart_decomp, (ilen / 1024 / 1024) / ((float) (timeend_decomp - timestart_decomp) / 1000));


	} else {

		sw842_compress(inH, ilen, compressedH, &olen);
		printf("copying compressed data to device\n");
		cuda_error = hipMemcpy(compressedD, compressedH, olen, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);
        printf("starting with device-based decompression\n");
        cuda842_decompress<<<1,1>>>(compressedD, decompressedD, 1);
        printf("copying decompressed data back to the host\n");
		cuda_error = hipMemcpy(decompressedH, decompressedD, dlen, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        CHECK_ERROR(cuda_error);

	}
	
	if (memcmp(inH, decompressedH, ilen) == 0) {
		printf("Compression- and decompression was successful!\n");
	} else {
		fprintf(stderr, "FAIL: Decompressed data differs from the original input data.\n");
	}
	hipHostFree(inH);
	hipHostFree(compressedH);
	hipHostFree(decompressedH);

	hipFree(compressedD);
	hipFree(decompressedD);
	hipStreamDestroy(stream);
	printf("\n\n");
	return 0;
}
